
#include <hip/hip_runtime.h>
extern "C" {

// ReLU 已有，这里补充其他激活函数

__global__ void sigmoid_forward(const float* input, float* output, int n, int inplace) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float v = input[idx];
        float res = 1.0f / (1.0f + expf(-v));
        if (inplace) {
            ((float*)input)[idx] = res;
        } else {
            output[idx] = res;
        }
    }
}

__global__ void tanh_forward(const float* input, float* output, int n, int inplace) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float v = input[idx];
        float res = tanhf(v);
        if (inplace) {
            ((float*)input)[idx] = res;
        } else {
            output[idx] = res;
        }
    }
}

__global__ void leakyrelu_forward(const float* input, float* output, int n, float alpha, int inplace) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float v = input[idx];
        float res = (v > 0) ? v : alpha * v;
        if (inplace) {
            ((float*)input)[idx] = res;
        } else {
            output[idx] = res;
        }
    }
}

__global__ void elu_forward(const float* input, float* output, int n, float alpha, int inplace) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float v = input[idx];
        float res = (v > 0) ? v : alpha * (expf(v) - 1.0f);
        if (inplace) {
            ((float*)input)[idx] = res;
        } else {
            output[idx] = res;
        }
    }
}

void launchSigmoid(const float* input, float* output, int n, int inplace) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    sigmoid_forward<<<blocks, threads>>>(input, output, n, inplace);
}

void launchTanh(const float* input, float* output, int n, int inplace) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    tanh_forward<<<blocks, threads>>>(input, output, n, inplace);
}

void launchLeakyReLU(const float* input, float* output, int n, float alpha, int inplace) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    leakyrelu_forward<<<blocks, threads>>>(input, output, n, alpha, inplace);
}

void launchELU(const float* input, float* output, int n, float alpha, int inplace) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    elu_forward<<<blocks, threads>>>(input, output, n, alpha, inplace);
}

} // extern "C"
